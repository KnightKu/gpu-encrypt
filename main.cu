#include "hip/hip_runtime.h"
/*
   This program is a modified version of the Hill cipher. It generates an nxn
   encryption and decryption key, and reads in a text file as an argument 
   which is stored in an nxn array. The data in the array is expanded to form 
   an nxnx8 array of bits -- each character from the file is expanded to its 
   ascii/binary form and stored in a "bit plane." Each bit plane is then 
   encrypted/decrypted using the encryption/decryption key. Multiplication 
   algorithms use are Square-Matrix-Multiply and Strassen-Recursive.

*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <stdint.h>

#include "matrixmul_kernel.cu"

extern int** identi(int);
extern int** decryptKey(int**, int);

int** allocate(int size)
{
	int i;
	int** m= (int**)malloc(size * sizeof(int*));
	if(m == NULL)
	{
		printf("Out of memory");
		exit(0);
	}
	for(i=0; i<size; i++)
	{
		m[i] = (int*)malloc(size * sizeof(int));
		if(m[i] == NULL)
		{
			printf("out of memory");
			exit(0);
		}
	}
	return m;
}

void deallocate(int** m, int s)
{
	int i;
	for(i=0; i<s; i++)
	{
		free(m[i]);
	}
	free(m);
}

int** add(int** a, int** b, int s)
{
	int i,j;
	int** m = allocate(s);
	for(i=0; i< s; i++)
	{
		for(j=0; j< s; j++)
		{
			m[i][j] = (a[i][j] + b[i][j])%2;	
		}
	}
	return m;
}

int** subtract(int** a, int** b, int s)
{
	int i,j;
	int** m = allocate(s);
	for(i=0; i < s; i++)
	{
		for(j=0; j < s; j++)
		{
			m[i][j] = (a[i][j] - b[i][j])%2;
		}
	}
	return m;
}

int** multiply(int** a, int** b, int size)
{
	int i,j;
	int** c = allocate(size);
	
	if(size == 1)
	{
		c[0][0] = (a[0][0] * b[0][0])%2;
		return c;
	}

	if(size <= 2)
	{
		int a11,a12,a21,a22,b11,b12,b21,b22;	
		a11 = a[0][0];
		a12 = a[0][1];
		a21 = a[1][0];
		a22 = a[1][1];
		b11 = b[0][0];
		b12 = b[0][1];
		b21 = b[1][0];
		b22 = b[1][1];
		
		c[0][0] = (a11*b11 + a12*b21)%2;
        	c[0][1] = (a11*b12 + a12*b22)%2;
        	c[1][0] = (a21*b11 + a22*b21)%2;
		c[1][1] = (a21*b12 + a22*b22)%2;
        	return c;
	}

	size = size/2;

	int** A11 = allocate(size);
	int** A12 = allocate(size);
	int** A21 = allocate(size);
	int** A22 = allocate(size);
	int** B11 = allocate(size);
	int** B12 = allocate(size);
	int** B21 = allocate(size);
	int** B22 = allocate(size);

	for(i=0;i<size;i++)
	{
		for(j=0;j<size;j++)
		{
			A11[i][j] = a[i][j];	
			A12[i][j] = a[i][j+size];
			A21[i][j] = a[i+size][j];
			A22[i][j] = a[i + size][j + size];
			B11[i][j] = b[i][j];
			B12[i][j] = b[i][j + size];
			B21[i][j] = b[i + size][j];
			B22[i][j] = b[i + size][j + size];
		}
	}
	
	int** S1 = subtract(B12,B22,size);
	int** S2 = add(A11,A12, size);
	int** S3 = add(A21,A22, size);
	int** S4 = subtract(B21,B11, size);
	int** S5 = add(A11,A22, size);
	int** S6 = add(B11,B22, size);
	int** S7 = subtract(A12,A22, size);
	int** S8 = add(B21,B22, size);
	int** S9 = subtract(A11,A21, size);
	int** S10 = add(B11,B12, size);

	int** P1 = multiply(A11, S1, size);
	int** P2 = multiply(S2, B22, size);
	int** P3 = multiply(S3, B11, size);
	int** P4 = multiply(A22, S4, size);
	int** P5 = multiply(S5, S6, size);
	int** P6 = multiply(S7, S8, size);
	int** P7 = multiply(S9, S10,size);

	int** c11 = subtract(add(P5,P4,size), add(P2,P6,size), size);
	int** c12 = add(P1,P2,size);
	int** c21 = add(P3,P4,size);
	int** c22 = subtract(add(P5,P1,size), subtract(P3,P7,size), size);
	
	int** temp = add(P5,P4,size);
	int** temp2 = add(P2,P6, size);

	for(i=0; i< size; i++)
	{
		for(j=0; j< size; j++)
		{
			c[i][j] = abs(c11[i][j] % 2);			
			c[i][j+size] = abs(c12[i][j] % 2);
			c[i+size][j] = abs(c21[i][j] % 2);
			c[i+size][j+size] = abs(c22[i][j] % 2);
		}
	}

	deallocate(A11, size);
	deallocate(A12, size);
	deallocate(A21, size);
	deallocate(A22, size);
	deallocate(B11, size);
	deallocate(B12, size);
	deallocate(B21, size);
	deallocate(B22, size);
	deallocate(c11, size);
	deallocate(c12, size);
	deallocate(c21, size);
	deallocate(c22, size);
	deallocate(P1, size);
	deallocate(P2, size);
	deallocate(P3, size);
	deallocate(P4, size);
	deallocate(P5, size);
	deallocate(P6, size);
	deallocate(P7, size);
	deallocate(S1, size);
	deallocate(S2, size);
	deallocate(S3, size);
	deallocate(S4, size);
	deallocate(S5, size);
	deallocate(S6, size);
	deallocate(S7, size);
	deallocate(S8, size);
	deallocate(S9, size);
	deallocate(S10, size);
	deallocate(temp, size);
	deallocate(temp2, size);
	return c;
}

int** squareMatrixMultiply(int** a, int** b, int size)
{
	int i,j,k;
	int** c = allocate(size);
	for(i = 0; i < size; i++)
	{
		for(j = 0; j < size; j++)
		{
			c[i][j] = 0;	
			for(k = 0; k < size; k++)
			{
				c[i][j] = abs(c[i][j] + (a[i][k] * b[k][j]))%2;
			}
		}
	}
	return c;
}

int** generateKey(int n)
{
	int i,j;
	int** k = (int**)malloc(n * sizeof(int*));
	for(i = 0; i < n; i++)
	{
		k[i] = (int*)malloc(n * sizeof(int));
	}
	for(i = 0; i < n; i++)
	{
		for(j = 0; j < n; j++)
		{
			k[i][j] = rand() % 2;
		}
	}
	return(k);
}

void strassenRecursive(int input, int*** nd, int yaxis, int*** em, int** key, int count)
{
	int i,j,n,k,m;
	srand(time(NULL));
	hipEvent_t start;
	hipEvent_t stop;
	float msecTotal;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	n = input;
	int** b = (int**)malloc(n * sizeof(int*));
	int** c;
	for(i=0; i<n; i++)
	{
		b[i] = (int*)malloc(n * sizeof(int));
	}
	int v = 0;
	hipEventRecord(start,NULL);
	for(k = 0; k < 8; k++)
	{
		for(m = 0; m < yaxis/n; m++)
		{
			for(i=0; i < n; i++)
			{
				for(j = 0; j < n; j++)
				{
					b[i][j] = nd[i][j+(m*n)][k];
					v++;
				}
			}
			/* Multiply encryption matrix for each nxn matrix in the bitplane */
			if(count == 0)
				c = multiply(key,b,n);
			else if(count == 1)
				c = squareMatrixMultiply(key, b, n);
			for(i=0; i<n; i++)
			{
				for(j = 0; j < n; j++)
				{					/* Copy encrypted values to new 3D array */

					em[i][j+(m*n)][k] = c[i][j];
				}
			}
			deallocate(c, n);
		}
	}
	hipEventRecord(stop,NULL);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);

	printf("%.3fms\n", msecTotal);
	deallocate(b, n);
}

void retrieveFromBinary(int*** nd, int size, int yaxis, int ex)
{
	int i,j,k;
	char c;
	int count;
	for(i = 0; i < size; i++)
	{
		for(j = 0; j < yaxis; j++)
		{
			for(k = 7; k >= 0; k--)
			{
				c <<= 1;
	 			c += nd[i][j][k];
			}
			count++;
			if(ex == 0)
				printf("%c", c);
			else
				printf("%d", c);
		}
	}
	printf("\n\n");
}

void storeToBinary(int letter, int index, int height, int*** nd, int size)
{
	int i,j,k;
	for(i = 7; i >= 0; i--)
	{
		j = letter >> i;
		if(j & 1)
			k = 1;
		else
			k = 0;
		nd[index%size][height][i] = k;
	}
	
}

int compare(int** A, int size)
{
	int** identity; 
	identity = (int**)(intptr_t)identi(size);
	int i,j;
	for(i = 0; i < size; i++)	
	{
		for(j = 0; j < size; j++)
		{
			if(!(A[i][j] == identity[i][j]))				
			{
				deallocate(identity, size);
				return(0);
			}
		}
	}
	deallocate(identity, size);
	return(1);
}

void hillCipher(int size, char* fname, int** key, int** dkey, int count)
{
	FILE* file = fopen(fname, "r");
	char *s;
	long bufsize;
	int yaxis;
	size_t length;
        if(fseek(file, 0L, SEEK_END) == 0)
        {
                bufsize = ftell(file);
                if(bufsize == -1)
                {
                        printf("Error in buff");
                        exit(0);
                }
                s = (char*)malloc(sizeof(char) * (bufsize + 1));
                if(fseek(file, 0L, SEEK_SET) != 0)
                {
                        printf("Error in seek");
                }
                length = fread(s, sizeof(char), bufsize, file);
                if(length == 0)
                {
                        fputs("Error", stderr);
                }
                else
                {
                        s[++length] = '\0';
                }
        }
	int x,y;
	int z = 0;
	int*** dm = (int***)malloc(size * sizeof(int**));
	int*** enMatrix = (int***)malloc(size * sizeof(int**));
	int*** dMatrix = (int***)malloc(size * sizeof(int**));
	/* determine the number of rows */
	yaxis = length/size;
	while(yaxis%size != 0) //fill in rows with 0's until divide evenly
		yaxis += 1;
	for(x = 0; x < size; x++)
	{
		dm[x] = (int**)malloc(yaxis * sizeof(int*));
		enMatrix[x] = (int**)malloc(yaxis * sizeof(int*));
		dMatrix[x] = (int**)malloc(yaxis * sizeof(int*));
		for(y = 0; y < yaxis; y++)
		{
			dm[x][y] = (int*)malloc(8 * sizeof(int));
			enMatrix[x][y] = (int*)malloc(8 * sizeof(int));
			dMatrix[x][y] = (int*)malloc(8 * sizeof(int));
			if(z < (int)length)
			{
				storeToBinary((int)s[z], x, y, dm, size);
				printf("%c", s[z]);
			}
			else
			{
				storeToBinary(0, x, y, dm, size);
			}
			z++;
		}
	}
	printf("Encrypt\n");
	strassenRecursive(size, dm, yaxis, enMatrix, key, count);
	retrieveFromBinary(enMatrix, size, yaxis, 1);
	printf("Decrypt\n");
	strassenRecursive(size, enMatrix, yaxis, dMatrix, dkey, count);
	retrieveFromBinary(dMatrix, size, yaxis, 0);
	free(dm);
	free(enMatrix);
	free(dMatrix);
}

int main(int argc, char *argv[])
{
	srand(time(NULL));
	int k, det, count,ka;
	int i,j,n,m;
	int cuda = 0;
//Cuda variables
	unsigned int mem_size_A;
	unsigned int mem_size_B;
	unsigned int mem_size_C;
	hipEvent_t start;
	hipEvent_t stop;
	float msecTotal;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int** key;
	int** test;
	char* f = argv[1];
	int input[7] = {8,16,32,64,128,256,512};
	int** dkey;
	for(count = 0; count < 1; count++)
	{
		if(count == 0){
			printf("Strassen Times\n");
		}
		else if(count == 1){	
			printf("Square Matrix Multiply\n");
		}
		else
			printf("Cuda Times (encryption, decryption) are\n");
	for(k = 0; k < 1; k++){
		det = 0;
		while(det == 0){
			key = generateKey(input[k]);
			dkey = (int**)(intptr_t)decryptKey(key, input[k]);
			test = squareMatrixMultiply(key, dkey, input[k]);
			det = compare(test, input[k]);
			if(det == 0){
				deallocate(key, input[k]);
				deallocate(dkey, input[k]);
				deallocate(test, input[k]);
			}
		}
		if(count < 2){
	//		hipEventRecord(start, NULL);
			hillCipher(input[k], f, key, dkey,count);
	//		hipEventRecord(stop, NULL);
	//		hipEventSynchronize(stop);
	//		hipEventElapsedTime(&msecTotal, start, stop);
	//		printf("%.3fms\n", msecTotal);
			deallocate(key, input[k]);
			deallocate(dkey, input[k]);
			deallocate(test, input[k]);
		}
		else{
			FILE* file = fopen(f, "r");
			char *s;
			long bufsize;
			int yaxis;
			size_t length;
			if(fseek(file, 0L, SEEK_END) == 0)
			{
				bufsize = ftell(file);
				if(bufsize == -1)
				{
				        printf("Error in buff");
				        exit(0);
				}
				s = (char*)malloc(sizeof(char) * (bufsize + 1));
				if(fseek(file, 0L, SEEK_SET) != 0)
				{
				        printf("Error in seek");
				}
				length = fread(s, sizeof(char), bufsize, file);
				if(length == 0)
				{
				        fputs("Error", stderr);
				}
				else
				{
				        s[++length] = '\0';
				}
			}
			int x,y;
			int z = 0;
			int*** dm = (int***)malloc(input[k] * sizeof(int**));
			int*** enMatrix = (int***)malloc(input[k] * sizeof(int**));
			int*** dMatrix = (int***)malloc(input[k] * sizeof(int**));
			yaxis = length/input[k];
			mem_size_A = sizeof(float) * input[k] * input[k];
			fclose(file);
			while(yaxis%input[k] != 0) //fill in rows with 0's until divide evenly
				yaxis += 1;
			mem_size_B = sizeof(float) * input[k] * yaxis * 8;
			mem_size_C = sizeof(float) * input[k] * yaxis * 8;
			for(x = 0; x < input[k]; x++)
			{
				dm[x] = (int**)malloc(yaxis * sizeof(int*));
				enMatrix[x] = (int**)malloc(yaxis * sizeof(int*));
				dMatrix[x] = (int**)malloc(yaxis * sizeof(int*));
				for(y = 0; y < yaxis; y++)
				{
					dm[x][y] = (int*)malloc(8 * sizeof(int));
					enMatrix[x][y] = (int*)malloc(8 * sizeof(int));
					dMatrix[x][y] = (int*)malloc(8 * sizeof(int));
					if(z < (int)length)
					{
						storeToBinary((int)s[z], x, y, dm, input[k]);
					}
					else
					{
						storeToBinary(0, x, y, dm, input[k]);
					}
					z++;
				}
			}
//FIRST STRASSEN
				n = input[k];
			//	int** b = (int**)malloc(n * sizeof(int*));
			//	int** c;
				float* d_A, *d_B, *d_C;
				hipMalloc((void**) &d_A, mem_size_A);
				hipMalloc((void**) &d_B, mem_size_B);
				hipMalloc((void**) &d_C, mem_size_C);
				float* h_A = (float*)malloc(mem_size_A);
				float* h_B = (float*)malloc(mem_size_B);
				float* h_C = (float*)malloc(mem_size_C);
				for(i = 0; i < n; i++)
				{
				//	b[i] = (int*)malloc(n * sizeof(int));
				}
//Encryption
				for(ka = 0; ka < 8; ka++)
				{
					for(m = 0; m < yaxis; m++)
					{
						for(i=0; i < n; i++)
						{
							h_B[i+(m*n)+(ka*(n*yaxis))] = dm[i][m][ka];
						}
						//	c = multiply(key,b,n);
						for(i=0; i<n; i++)
						{
							for(j = 0; j < n; j++){

						//		enMatrix[i][j+(m*n)][ka] = c[i][j];
							}
						}
					//	deallocate(c, n);
					}
				}
				for(i = 0; i < n; i++)
				{
					for(j = 0; j < n; j++)
					{
						h_A[j+(i*n)] = key[i][j];
					}
				}
				hipEventRecord(start,NULL);
				matrixMul<<<input[k],16>>>(d_C, d_A, d_B, input[k], input[k]);
				hipDeviceSynchronize();
				hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
				hipEventRecord(stop, NULL);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&msecTotal, start, stop);
				printf("Encrypt: %.3fms\n", msecTotal);
				//retrieveFromBinary(enMatrix, input[k], yaxis);
//SECOND STRASSEN
				n = input[k];
			//	int** ba = (int**)malloc(n * sizeof(int*));
			//	int** ca;
				for(i = 0; i < n; i++)
				{
			//		ba[i] = (int*)malloc(n * sizeof(int));
				}
				for(ka = 0; ka < 8; ka++)
				{
					for(m = 0; m < yaxis; m++)
					{
						for(i=0; i < n; i++)
						{
							h_B[i+(m*n)+(ka*(n*yaxis))] = enMatrix[i][m][ka];
						}
					}
				}
				for(i = 0; i < n; i++)
				{
					for(j = 0; j < n; j++)
					{
						h_A[j+(i*n)] = dkey[i][j];
					}
				}
				hipEventRecord(start,NULL);
				matrixMul<<<input[k],16>>>(d_C, d_A, d_B, input[k], input[k]);
				hipDeviceSynchronize();
				hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
//				deallocate(ba, n);*/
			//	retrieveFromBinary(dMatrix, input[k], yaxis);
				hipEventRecord(stop,NULL);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&msecTotal, start, stop);
				printf("Decrypt: %.3fms\n", msecTotal);
				free(dm);
				free(enMatrix);
				free(dMatrix);
				free(h_A);
				free(h_B);
				free(h_C);
				hipFree(d_A);
				hipFree(d_B);
				hipFree(d_C);
		}
	}
	}
return 0;
}
